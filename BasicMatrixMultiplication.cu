#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
#include "BasicMatrixMultiplication_Template.h"
using namespace std;

#define BLOCK_SIZE 16


//Compute C=A*B
// Serial implementation for running on CPU using a single thread.
void MatrixMultiplyCpu(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	//@@ Insert Your Code Here for the CPU Function to Compute Matrix Maltiply
	float sum;
	for (int row = 0; row < numARows; row++) {
		for (int col = 0; col < numBColumns; col++) {
			sum = 0;
			for (int i = 0; i < numAColumns; i++) {
				sum += A [row*numAColumns+i] * B[i*numBColumns+ col];
			}

			C[row*numBColumns+col] = sum;
			
		}
	}
}


//GPU Kernel for Basic Matrix Multiplication
__global__ void BasixMatrixMultiply(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	//@@ Insert Your Code Here for the CUDA Kernel for Basic Matrix Multiply
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	float sum = 0;
	if (row < numARows && col < numBColumns) {
		for (int i = 0; i < numAColumns; i++) {
			sum += A[row *numAColumns+ i] * B[i * numBColumns + col];
		}
		C[row*numBColumns+col] = sum;
	}
}



int main(void)
{
	int numARows=960; // number of rows in the matrix A
	int numAColumns=640; // number of columns in the matrix A
	int numBRows=640; // number of rows in the matrix B
	int numBColumns=800; // number of columns in the matrix B
	
	int numCRows; // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set this)
	
	//@@ Insert Your Code Here to Set numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;

	//Allocate the host memory for the input and output matrices
	float *h_A = (float *)malloc(sizeof(float)*numARows*numAColumns);
	float *h_B = (float *)malloc(sizeof(float)*numBRows*numBColumns);
	float *h_C = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	float *h_C_CPU = (float *)malloc(sizeof(float)*numCRows*numCColumns);


	//Random Initialize Matrix A. 
	//There are several ways to do this, such as making functions for manual input or using random numbers. 
	//In this case, we simply use a for loop to fill the cells with trigonometric values of the indices:
	// Set the Seed for the random number generator rand() 
	//srand(clock());
	for (int i=0; i<numARows; i++)
	{
		for (int j=0; j<numAColumns; j++)
		{
			//h_A[i*numAColumns+j]=(float)rand() /(float)(RAND_MAX)*4.0;
			h_A[i*numAColumns+j]=sin(i);
		}
	}

	//Random Initialize Matrix B
	for (int i=0; i<numBRows; i++)
	{
		for (int j=0; j<numBColumns; j++)
		{
			//h_B[i*numBColumns+j]=(float)rand() /(float)(RAND_MAX) *4.0;
			h_B[i*numBColumns+j]=cos(j);

		}
	}

	//Allocate memory on the device for input and output matrices and record the needed time
	float *d_A, *d_B, *d_C;
	hipError_t	err = hipSuccess;
	GpuTimer timer;
	timer.Start();

	//@@Insert Your Code Here to allocate memory for d_A, d_B, d_C
	err = hipMalloc((void	**)&d_A, sizeof(float)*numARows*numAColumns);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed	to allocate device memory for vecotr A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void	**)&d_B, sizeof(float)*numBRows*numBColumns);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed	to allocate device memory for vecotr A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void	**)&d_C, sizeof(float)*numCRows*numCColumns);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed	to allocate device memory for vecotr A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	timer.Stop();
	printf("Time to allocate memory on the device is: %f msecs.\n", timer.Elapsed());



	//Copy the input matrices A and B from the host to the device and record the needed time
	GpuTimer timer1;
	timer1.Start();
	
	//@@ Insert Your Code Here to copy matrices A and B from Host to Device
	hipMemcpy(d_A, h_A, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);

	timer1.Stop();
	printf("Time to copy the Matrix from the host to the device is: %f msecs.\n", timer1.Elapsed());


	//Do the Processing on the GPU
	//@@ Insert Kernel Execution Configuration Parameters
	dim3 dimGrid( ceil(numBColumns/(float)BLOCK_SIZE) , ceil(numAColumns/(float)BLOCK_SIZE), 1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

	

	//Invoke the BasicMatrixMultiply kernel and record the needed time for its execution
	GpuTimer timer2;
	timer2.Start();
	//@@ Insert Your Code Here for Kernel Invocation
	BasixMatrixMultiply <<<dimGrid, dimBlock >>> (d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	timer2.Stop();
	printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

	//Copy resulting matrix from device to host and record the needed time
	GpuTimer timer3;
	timer3.Start();
	//@@ Insert Your Code Here to Copy the resulting Matrix d_C from device to the Host h_C
	hipMemcpy(h_C, d_C, sizeof(float)*numCColumns*numCRows, hipMemcpyDeviceToHost);
	timer3.Stop();
	printf("Time to copy the resulting Matrix from the device to the host is: %f msecs.\n", timer3.Elapsed());


	//Do the Processing on the CPU
	clock_t begin = clock();
	//@@ Insert Your Code Here to call the CPU function MatrixMultiplyCpu where the resulting matrix is h_C_CPU
	MatrixMultiplyCpu(h_A, h_B, h_C_CPU, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	printf("Implemented CPU serial code ran in: %f msecs.\n", time_spent);

	//Verify Results Computed by GPU and CPU
	for (int i=0; i<numCRows; i++)
		for (int j=0; j<numCColumns; j++)
		
			if (fabs(h_C_CPU[i*numCColumns+j] - h_C[i*numCColumns+j]) > 1e-2)
			{
				fprintf(stderr, "Result verification failed at element (%d,%d)!\n", i,j);
				exit(EXIT_FAILURE);
			}
	printf("Test PASSED\n");


	//Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_CPU);

	//Free device memory
	//@@ Insert Your Code Here to Free Device Memory
	hipFree(d_A); hipFree(d_B); hipFree(d_C);

	return 0;

}