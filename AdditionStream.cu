#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 256
#define SegLength 1024*10



//CUDA Kernel Device code
//Computes the element-wise vector addition of A and B into C: C[i] = A[i] + B[i].
//The 3 vectors have the same number of elements numElements.
__global__ void vectorAdd( float *A,  float *B, float *C, int numElements)
	
{
	//@@ Insert  your code here to implement vector addition where each thread performs one addition.
	int index = threadIdx.x + blockDim.x*blockIdx.x;
	if (index < numElements)
		C[index] = A[index] + B[index];
	
}

/**
* Host main routine
*/
int main(void)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	float EPS = 0.0001;
	int numElements = 4096000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	//Implement Vector Addition without using CUDA Streams
	// Allocate the host input vector A
	float *h_A = (float *)malloc(size);

	// Allocate the host input vector B
	float *h_B = (float *)malloc(size);

	// Allocate the host output vector C
	float *h_C = (float *)malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = float(i);
		h_B[i] = 1/(i+EPS);
	}


	GpuTimer timer;
	timer.Start();
	// Allocate the device input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector B
	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector C
	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	
	timer.Stop();
	printf("Time for vector addition implemenation without using CUDA streams: %f msecs.\n", timer.Elapsed());


	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
		if (fabs((h_A[i] + h_B[i]) - h_C[i]) > 1e-5)
		{
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");

	// Free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	hipDeviceProp_t prop;
	int dev_count;

	hipGetDeviceCount(&dev_count);
	for (int i = 0; i < dev_count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		if (!prop.deviceOverlap)
		{
			printf("Unable to handle overlap. Exiting...\n");
			return 0;
		}
	}
	hipStream_t stream1, stream2, stream3, stream4;

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);

	//Implement Vector Addition Using CUDA Streams
	
	GpuTimer timer1;
	timer1.Start();
	
	//@@ Insert your code here to implement Vector Addition using streams and Time your implementation.Use the already allocated and initialized host arrays
	//h_A, h_B, and h-C;
	hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
	hipHostAlloc((void **)&h_B, size, hipHostMallocDefault);
	hipHostAlloc((void **)&h_C, size, hipHostMallocDefault);

	float *d_A0 = NULL; float *d_A1 = NULL; float *d_A2 = NULL; float *d_A3 = NULL;
	hipMalloc((void **)&d_A0, size); hipMalloc((void **)&d_A1, size); hipMalloc((void **)&d_A2, size); hipMalloc((void **)&d_A3, size);

	float *d_B0 = NULL; float *d_B1 = NULL; float *d_B2 = NULL; float *d_B3 = NULL; 
	hipMalloc((void **)&d_B0, size); hipMalloc((void **)&d_B1, size); hipMalloc((void **)&d_B2, size); hipMalloc((void **)&d_B3, size); 

	float *d_C0 = NULL; float *d_C1 = NULL; float *d_C2 = NULL; float *d_C3 = NULL;
	hipMalloc((void **)&d_C0, size); hipMalloc((void **)&d_C1, size); hipMalloc((void **)&d_C2, size); hipMalloc((void **)&d_C3, size); 

	for (int i = 0; i < numElements; i += SegLength * 4) {
		hipMemcpyAsync(d_A0, h_A + i, SegLength * sizeof(float), hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(d_B0, h_B + i, SegLength * sizeof(float), hipMemcpyHostToDevice, stream1);
		
		hipMemcpyAsync(d_A1, h_A + i + SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(d_B1, h_B + i + SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream2);

		hipMemcpyAsync(d_A2, h_A + i + 2 * SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream3);
		hipMemcpyAsync(d_B2, h_B + i + 2 * SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream3);

		hipMemcpyAsync(d_A3, h_A + i + 3 * SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream4);
		hipMemcpyAsync(d_B3, h_B + i + 3 * SegLength, SegLength * sizeof(float), hipMemcpyHostToDevice, stream4);

		vectorAdd <<< blocksPerGrid, threadsPerBlock, 0, stream1 >>> (d_A0, d_B0, d_C0, numElements);
		vectorAdd <<< blocksPerGrid, threadsPerBlock, 0, stream2 >>> (d_A1, d_B1, d_C1, numElements);
		vectorAdd <<< blocksPerGrid, threadsPerBlock, 0, stream3 >>> (d_A2, d_B2, d_C2, numElements);
		vectorAdd <<< blocksPerGrid, threadsPerBlock, 0, stream4 >>> (d_A3, d_B3, d_C3, numElements);

		hipMemcpyAsync(h_C + i, d_C0, SegLength * sizeof(float), hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(h_C + i + SegLength, d_C1, SegLength * sizeof(float), hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(h_C + i + 2 * SegLength, d_C2, SegLength * sizeof(float), hipMemcpyDeviceToHost, stream3);
		hipMemcpyAsync(h_C + i + 3 * SegLength, d_C3, SegLength * sizeof(float), hipMemcpyDeviceToHost, stream4);

	}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);
	hipStreamSynchronize(stream4);
	
	timer1.Stop();
	printf("Time for vector addition implementation using CUDA streams: %f msecs.\n", timer1.Elapsed());

	//@@Insert your code to free device memory and streams
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipFree(d_A0); hipFree(d_A1); hipFree(d_A2); hipFree(d_A3); 
	hipFree(d_B0); hipFree(d_B1); hipFree(d_B2); hipFree(d_B3);
	hipFree(d_C0); hipFree(d_C1); hipFree(d_C2); hipFree(d_C3); 
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);
	hipStreamDestroy(stream4);

	printf("Done\n");

	return 0;
}

